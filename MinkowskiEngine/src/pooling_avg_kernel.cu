#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA Corporation.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#ifndef GPU_POOLING_AVG
#define GPU_POOLING_AVG

#include <hipsparse.h>
#include <limits>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

#include <thrust/functional.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include "allocators.cuh"
#include "pooling_avg_kernel.cuh"
#include "utils.hpp"

namespace minkowski {

template <typename Dtype>
__global__ void fill(const int n, Dtype *in_feat, Dtype val) {
  CUDA_KERNEL_LOOP(index, n) { in_feat[index] = val; }
}

template <typename Dtype>
__global__ void col2row_major(const int n, const int nrows, const int ncols,
                              const Dtype *colA, Dtype *rowA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index % nrows;
    j = index / nrows;
    rowA[i * ncols + j] = colA[index];
  }
}

template <typename Dtype>
__global__ void col2row_major_with_div(const int n, const int nrows,
                                       const int ncols,
                                       const Dtype *num_nonzero,
                                       const Dtype *colA, Dtype *rowA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index % nrows;
    j = index / nrows;
    if (num_nonzero[i] >= 1) {
      rowA[i * ncols + j] = colA[index] / num_nonzero[i];
    } else {
      rowA[i * ncols + j] = colA[index];
    }
  }
}

template <typename Itype, typename Dtype>
__global__ void
unique_row2num_nonzero(const int n, Dtype *__restrict__ d_num_nonzero,
                       const Itype *__restrict__ unique_row_ptr,
                       const Dtype *__restrict__ reduced_val_ptr) {
  CUDA_KERNEL_LOOP(index, n) {
    d_num_nonzero[unique_row_ptr[index]] = reduced_val_ptr[index];
  }
}

template <typename Dtype, typename Itype>
__global__ void set_gradient(const int n, const Dtype *d_grad_out,
                             Dtype *d_grad_in, const Itype *out_index,
                             int nchannel) {
  CUDA_KERNEL_LOOP(index, n) {
    atomicAdd(&d_grad_in[out_index[index]], d_grad_out[index]);
  }
}

template <typename Dtype, typename Itype>
__global__ void
set_gradient_nonzero(const int n, const Dtype *d_grad_out, Dtype *d_grad_in,
                     int nchannel, const Itype *in_map, const Itype *out_map) {
  CUDA_KERNEL_LOOP(index, n) {
    int nrow = index / nchannel;
    int ch = index % nchannel;
    atomicAdd(&d_grad_in[in_map[nrow] * nchannel + ch],
              d_grad_out[out_map[nrow] * nchannel + ch]);
  }
}

template <typename Dtype, typename Itype>
__global__ void
set_gradient_nonzero_avg(const int n, const Dtype *d_grad_out, Dtype *d_grad_in,
                         int nchannel, const Dtype *d_num_nonzero,
                         const Itype *in_map, const Itype *out_map) {
  CUDA_KERNEL_LOOP(index, n) {
    int nrow = index / nchannel;
    int ch = index % nchannel;
    int curr_num_nonzero = d_num_nonzero[out_map[nrow]];
    if (curr_num_nonzero >= 1)
      atomicAdd(&d_grad_in[in_map[nrow] * nchannel + ch],
                d_grad_out[out_map[nrow] * nchannel + ch] / curr_num_nonzero);
  }
}

template <typename Dtype, typename Itype, typename ByteAllocator>
void NonzeroAvgPoolingForwardKernelGPU(
    Dtype const *d_in_feat,                                 //
    default_types::size_type const in_nrows,                //
    Dtype *d_out_feat,                                      //
    default_types::size_type const out_nrows,               //
    Dtype *d_num_nonzero,                                   //
    default_types::size_type const nchannel,                //
    gpu_kernel_map<Itype, ByteAllocator> const &kernel_map, //
    bool const use_avg,                                     //
    ByteAllocator &allocator,                               //
    hipsparseHandle_t cushandle, hipStream_t stream) {
  const Dtype alpha = 1;
  const Dtype beta = 0;
  static_assert(sizeof(Itype) == sizeof(int),
                "cusparse requires int type index");
  Dtype *d_ones, *d_coo_val, *d_tmp_out_feat;

  constexpr bool is_int32 = sizeof(Itype) == sizeof(int32_t);
  constexpr bool is_int64 = sizeof(Itype) == sizeof(int64_t);
  constexpr bool is_float32 = std::is_same<Dtype, float>::value;
  hipDataType cuda_data_type = is_float32 ? HIP_R_32F : HIP_R_64F;

  hipsparseSpMMAlg_t mm_alg;
#if defined(CUDART_VERSION) && (CUDART_VERSION < 10010)
  ASSERT(false, "spmm sparse-dense requires CUDA 10.1 or greater");
#elif defined(CUDART_VERSION) && (CUDART_VERSION >= 10010) &&                  \
    (CUDART_VERSION < 11000)
  mm_alg = HIPSPARSE_COOMM_ALG1;
  static_assert(is_int32, "int64 hipsparseSpMM requires CUDA 11.1 or greater");
#elif defined(CUDART_VERSION) && (CUDART_VERSION >= 11000)
  mm_alg = HIPSPARSE_SPMM_COO_ALG1;
  static_assert(is_int32 || is_int64, "Invalid index type");
#endif

  /* sparse mm prep */
  size_t const sparse_nnzs =
      kernel_map.in_maps.end() - kernel_map.in_maps.begin();
  static_assert(is_int32, "sort_coo supports int32");
  sort_coo_gpu<ByteAllocator>(cushandle, out_nrows, in_nrows, sparse_nnzs,
                              (int *)kernel_map.out_maps.begin(),
                              (int *)kernel_map.in_maps.begin(), allocator);

  // feature output
  d_tmp_out_feat =
      (Dtype *)allocator.allocate(nchannel * out_nrows * sizeof(Dtype));
  d_coo_val = (Dtype *)allocator.allocate(sparse_nnzs * sizeof(Dtype));
  fill<Dtype><<<GET_BLOCKS(sparse_nnzs, CUDA_NUM_THREADS), CUDA_NUM_THREADS, 0,
                stream>>>(sparse_nnzs, d_coo_val, (Dtype)1.);
  if (use_avg) {
    d_ones = (Dtype *)allocator.allocate(sparse_nnzs * sizeof(Dtype));
    fill<Dtype><<<GET_BLOCKS(sparse_nnzs, CUDA_NUM_THREADS), CUDA_NUM_THREADS,
                  0, stream>>>(sparse_nnzs, d_ones, (Dtype)1.);
  }

#ifdef DEBUG
  std::cout << "sparse_nnzs: " << sparse_nnzs << "\n";
  Itype *p_scr = (Itype *)std::malloc((sparse_nnzs)*2 * sizeof(Itype));
  CUDA_CHECK(hipMemcpy(p_scr, kernel_map.out_maps.begin(),
                        sparse_nnzs * sizeof(Itype), hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(p_scr + sparse_nnzs, kernel_map.in_maps.begin(),
                        sparse_nnzs * sizeof(Itype), hipMemcpyDeviceToHost));

  Itype step = std::max<Itype>(sparse_nnzs / 100, 1);
  Itype i = 0;
  for (; i < sparse_nnzs;) {
    std::cout << i;
    std::cout << " out_map: " << p_scr[i]
              << ", in_map: " << p_scr[i + sparse_nnzs] << "\n";
    i += step;
  }
  i -= step;
  for (; i < sparse_nnzs; ++i) {
    std::cout << i;
    std::cout << " out_map: " << p_scr[i]
              << ", in_map: " << p_scr[i + sparse_nnzs] << "\n";
  }
  std::free(p_scr);
  std::cout << "done printing\n";
#endif

  Itype *sorted_row_ptr =
      (Itype *)allocator.allocate(2 * (sparse_nnzs + 1) * sizeof(Itype));
  Itype *sorted_col_ptr = sorted_row_ptr + sparse_nnzs + 1;

  CUDA_CHECK(hipMemcpy(sorted_row_ptr, kernel_map.out_maps.begin(),
                        sparse_nnzs * sizeof(Itype), hipMemcpyDeviceToDevice));
  CUDA_CHECK(hipMemcpy(sorted_col_ptr, kernel_map.in_maps.begin(),
                        sparse_nnzs * sizeof(Itype), hipMemcpyDeviceToDevice));

  thrust::sort_by_key(thrust::device,               //
                      sorted_row_ptr,               // key begin
                      sorted_row_ptr + sparse_nnzs, // key end
                      sorted_col_ptr);

  //  +---------+ +---+
  //  | spm     | | i |
  //  +---------+ | n |
  //    in_nrows  |   |
  //              | F |
  //              |   |
  //              +---+
  //             nchannel
  size_t dim_i = out_nrows, dim_j = in_nrows, dim_k = nchannel;
  hipsparseSpMatDescr_t sparse_descr;
  hipsparseDnMatDescr_t dense_descr;
  hipsparseDnMatDescr_t result_descr;
  CUSPARSE_CHECK(
      hipsparseCreateCoo(&sparse_descr,             //
                        dim_i, dim_j, sparse_nnzs, //
                        sorted_row_ptr,            // rows
                        sorted_col_ptr,            // cols
                        d_coo_val,                 // coo vals
                        is_int32 ? HIPSPARSE_INDEX_32I : HIPSPARSE_INDEX_64I,
                        HIPSPARSE_INDEX_BASE_ZERO, cuda_data_type));

  CUSPARSE_CHECK(hipsparseCreateDnMat(&dense_descr,        //
                                     dim_k, dim_j, dim_k, //
                                     (void *)d_in_feat,   //
                                     cuda_data_type, HIPSPARSE_ORDER_COL));

  CUSPARSE_CHECK(hipsparseCreateDnMat(&result_descr,          //
                                     dim_i, dim_k, dim_i,    //
                                     (void *)d_tmp_out_feat, //
                                     cuda_data_type, HIPSPARSE_ORDER_COL));

  size_t buffer_size = 0;
  CUSPARSE_CHECK(hipsparseSpMM_bufferSize(
      cushandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
      (void *)&alpha, sparse_descr, dense_descr, (void *)&beta, result_descr,
      cuda_data_type, mm_alg, &buffer_size));

  // buffer size 0 for HIPSPARSE_SPMM_COO_ALG1, HIPSPARSE_SPMM_COO_ALG3,
  // HIPSPARSE_SPMM_COO_ALG4, and HIPSPARSE_SPMM_CSR_ALG1

  // WARNING: coo sorting must have been handled in the kernel map
  // decomposition.
  CUSPARSE_CHECK(hipsparseSpMM(cushandle,                        //
                              HIPSPARSE_OPERATION_NON_TRANSPOSE, //
                              HIPSPARSE_OPERATION_TRANSPOSE,     //
                              (void *)&alpha,                   //
                              sparse_descr, dense_descr,        //
                              (void *)&beta, result_descr,      //
                              cuda_data_type, mm_alg, &buffer_size));
#ifdef DEBUG
  CUDA_CHECK(hipStreamSynchronize(0));
#endif
  LOG_DEBUG("SPMM");

  if (use_avg) {
    Itype *unique_row_ptr =
        (Itype *)allocator.allocate(sparse_nnzs * sizeof(Itype));
    Dtype *reduced_val_ptr =
        (Dtype *)allocator.allocate(sparse_nnzs * sizeof(Dtype));

    // reduce by key
    auto end = thrust::reduce_by_key(thrust::device,               // policy
                                     sorted_row_ptr,               // key begin
                                     sorted_row_ptr + sparse_nnzs, // key end
                                     d_ones,         // value begin
                                     unique_row_ptr, // key out begin
                                     reduced_val_ptr // value out begin
    );

    int num_unique_keys = end.first - unique_row_ptr;
    LOG_DEBUG("Num unique keys:", num_unique_keys);

#ifdef DEBUG
    Itype *p_unique_row = (Itype *)std::malloc(num_unique_keys * sizeof(Itype));
    CUDA_CHECK(hipMemcpy(p_unique_row, unique_row_ptr,
                          num_unique_keys * sizeof(Itype),
                          hipMemcpyDeviceToHost));
    std::cout << "[" << PtrToString(p_unique_row, num_unique_keys) << "]\n";
    std::free(p_unique_row);

    Dtype *p_reduced_val =
        (Dtype *)std::malloc(num_unique_keys * sizeof(Dtype));
    CUDA_CHECK(hipMemcpy(p_reduced_val, reduced_val_ptr,
                          num_unique_keys * sizeof(Dtype),
                          hipMemcpyDeviceToHost));
    std::cout << "[" << PtrToString(p_reduced_val, num_unique_keys) << "]\n";
    std::free(p_reduced_val);
#endif
    // Copy the results to the correct output
    unique_row2num_nonzero<Itype, Dtype>
        <<<GET_BLOCKS(num_unique_keys, CUDA_NUM_THREADS), CUDA_NUM_THREADS, 0,
           stream>>>(num_unique_keys, d_num_nonzero, unique_row_ptr,
                     reduced_val_ptr);

    col2row_major_with_div<Dtype>
        <<<GET_BLOCKS(out_nrows * nchannel, CUDA_NUM_THREADS), CUDA_NUM_THREADS,
           0, stream>>>(out_nrows * nchannel, out_nrows, nchannel,
                        d_num_nonzero, d_tmp_out_feat, d_out_feat);
#ifdef DEBUG
    CUDA_CHECK(hipStreamSynchronize(0));
#endif
    LOG_DEBUG("col2row");

    // Delete tmp spaces
    allocator.deallocate((char *)unique_row_ptr, sparse_nnzs * sizeof(Itype));
    allocator.deallocate((char *)reduced_val_ptr, sparse_nnzs * sizeof(Dtype));
  } else {
    col2row_major<Dtype><<<GET_BLOCKS(out_nrows * nchannel, CUDA_NUM_THREADS),
                           CUDA_NUM_THREADS, 0, stream>>>(
        out_nrows * nchannel, out_nrows, nchannel, d_tmp_out_feat, d_out_feat);
  }

  CUSPARSE_CHECK(hipsparseDestroySpMat(sparse_descr));
  CUSPARSE_CHECK(hipsparseDestroyDnMat(dense_descr));
  CUSPARSE_CHECK(hipsparseDestroyDnMat(result_descr));

  allocator.deallocate((char *)d_coo_val, sparse_nnzs * sizeof(Dtype));
  allocator.deallocate((char *)d_tmp_out_feat,
                       nchannel * out_nrows * sizeof(Dtype));
  if (use_avg)
    allocator.deallocate((char *)d_ones, in_nrows * sizeof(Dtype));

  allocator.deallocate((char *)sorted_row_ptr,
                       2 * (sparse_nnzs + 1) * sizeof(Itype));
  CUDA_CHECK(hipStreamSynchronize(0));
}

// default_allocator
template void
NonzeroAvgPoolingForwardKernelGPU<float, uint32_t,
                                  detail::default_allocator<char>>(
    float const *d_in_feat,                   //
    default_types::size_type const in_nrows,  //
    float *d_out_feat,                        //
    default_types::size_type const out_nrows, //
    float *d_num_nonzero,                     //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<uint32_t, detail::default_allocator<char>> const
        &kernel_map, //
    bool const use_avg,
    detail::default_allocator<char> &allocator, //
    hipsparseHandle_t cushandle, hipStream_t stream);

template void
NonzeroAvgPoolingForwardKernelGPU<double, uint32_t,
                                  detail::default_allocator<char>>(
    double const *d_in_feat,                  //
    default_types::size_type const in_nrows,  //
    double *d_out_feat,                       //
    default_types::size_type const out_nrows, //
    double *d_num_nonzero,                    //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<uint32_t, detail::default_allocator<char>> const
        &kernel_map, //
    bool const use_avg,
    detail::default_allocator<char> &allocator, //
    hipsparseHandle_t cushandle, hipStream_t stream);

// c10_allocator
template void
NonzeroAvgPoolingForwardKernelGPU<float, uint32_t, detail::c10_allocator<char>>(
    float const *d_in_feat,                                                  //
    default_types::size_type const in_nrows,                                 //
    float *d_out_feat,                                                       //
    default_types::size_type const out_nrows,                                //
    float *d_num_nonzero,                                                    //
    default_types::size_type const nchannel,                                 //
    gpu_kernel_map<uint32_t, detail::c10_allocator<char>> const &kernel_map, //
    bool const use_avg,
    detail::c10_allocator<char> &allocator, //
    hipsparseHandle_t cushandle, hipStream_t stream);

template void NonzeroAvgPoolingForwardKernelGPU<double, uint32_t,
                                                detail::c10_allocator<char>>(
    double const *d_in_feat,                                                 //
    default_types::size_type const in_nrows,                                 //
    double *d_out_feat,                                                      //
    default_types::size_type const out_nrows,                                //
    double *d_num_nonzero,                                                   //
    default_types::size_type const nchannel,                                 //
    gpu_kernel_map<uint32_t, detail::c10_allocator<char>> const &kernel_map, //
    bool const use_avg,
    detail::c10_allocator<char> &allocator, //
    hipsparseHandle_t cushandle, hipStream_t stream);

// Backward
template <typename Dtype, typename Itype, typename ByteAllocator>
void NonzeroAvgPoolingBackwardKernelGPU(
    Dtype *d_grad_in_feat,                    //
    default_types::size_type const in_nrows,  //
    Dtype const *d_grad_out_feat,             //
    default_types::size_type const out_nrows, //
    Dtype const *d_num_nonzero,               //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<Itype, ByteAllocator> const &kernel_map, bool const use_avg,
    hipStream_t stream) {
  // d_grad_in_feat must be all set to 0

  size_t sparse_nnzs = kernel_map.in_maps.end() - kernel_map.in_maps.begin();

  if (use_avg) {
    set_gradient_nonzero_avg<Dtype>
        <<<GET_BLOCKS(sparse_nnzs * nchannel, CUDA_NUM_THREADS),
           CUDA_NUM_THREADS, 0, stream>>>(
            sparse_nnzs * nchannel, d_grad_out_feat, d_grad_in_feat, nchannel,
            d_num_nonzero, kernel_map.in_maps.cdata(),
            kernel_map.out_maps.cdata());
  } else {
    set_gradient_nonzero<Dtype>
        <<<GET_BLOCKS(sparse_nnzs * nchannel, CUDA_NUM_THREADS),
           CUDA_NUM_THREADS, 0, stream>>>(
            sparse_nnzs * nchannel, d_grad_out_feat, d_grad_in_feat, nchannel,
            kernel_map.in_maps.cdata(), kernel_map.out_maps.cdata());
  }

  CUDA_CHECK(hipDeviceSynchronize());
}

// default_allocator
template void
NonzeroAvgPoolingBackwardKernelGPU<float, uint32_t,
                                   detail::default_allocator<char>>(
    float *d_grad_in_feat,                    //
    default_types::size_type const in_nrows,  //
    float const *d_grad_out_feat,             //
    default_types::size_type const out_nrows, //
    float const *d_num_nonzero,               //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<uint32_t, detail::default_allocator<char>> const &kernel_map,
    bool const use_avg, hipStream_t stream);

template void
NonzeroAvgPoolingBackwardKernelGPU<double, uint32_t,
                                   detail::default_allocator<char>>(
    double *d_grad_in_feat,                   //
    default_types::size_type const in_nrows,  //
    double const *d_grad_out_feat,            //
    default_types::size_type const out_nrows, //
    double const *d_num_nonzero,              //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<uint32_t, detail::default_allocator<char>> const &kernel_map,
    bool const use_avg, hipStream_t stream);

// c10_allocator
template void NonzeroAvgPoolingBackwardKernelGPU<float, uint32_t,
                                                 detail::c10_allocator<char>>(
    float *d_grad_in_feat,                    //
    default_types::size_type const in_nrows,  //
    float const *d_grad_out_feat,             //
    default_types::size_type const out_nrows, //
    float const *d_num_nonzero,               //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<uint32_t, detail::c10_allocator<char>> const &kernel_map,
    bool const use_avg, hipStream_t stream);

template void NonzeroAvgPoolingBackwardKernelGPU<double, uint32_t,
                                                 detail::c10_allocator<char>>(
    double *d_grad_in_feat,                   //
    default_types::size_type const in_nrows,  //
    double const *d_grad_out_feat,            //
    default_types::size_type const out_nrows, //
    double const *d_num_nonzero,              //
    default_types::size_type const nchannel,  //
    gpu_kernel_map<uint32_t, detail::c10_allocator<char>> const &kernel_map,
    bool const use_avg, hipStream_t stream);

} // end namespace minkowski

#endif // end GPU_POOLING_AVG
