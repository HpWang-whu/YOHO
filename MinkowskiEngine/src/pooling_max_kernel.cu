#include "hip/hip_runtime.h"
/* Copyright (c) Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#ifndef GPU_POOLING_MAX_KERNEL
#define GPU_POOLING_MAX_KERNEL

#include <limits>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

#include <thrust/functional.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>

#include "allocators.cuh"
#include "gpu.cuh"
#include "pooling_max_kernel.cuh"
#include "utils.hpp"

template <typename Dtype, typename MaskItype>
__global__ void set_gradient(const size_t n, const Dtype *d_grad_out,
                             Dtype *d_grad_in, const MaskItype *in_index,
                             const Dtype unused_key) {
  CUDA_KERNEL_LOOP(index, n) {
    auto const queried_index = in_index[index];
    if (queried_index != unused_key)
      atomicAdd(&d_grad_in[in_index[index]], d_grad_out[index]);
  }
}

template <typename Dtype, typename MaskItype, typename MapItype>
__global__ void max_pool_reduced_pointer_kernel_gpu(
    const int N, const int out_nrows, const int nchannel, const int nmap,
    const Dtype *__restrict__ d_in_feat,    //
    Dtype *__restrict__ d_out_feat,         //
    MaskItype *__restrict__ d_max_index,    //
    const MapItype *__restrict__ d_in_map,  //
    const MapItype *__restrict__ d_out_map, //
    const MapItype *__restrict__ d_in_index_min) {
  // N == nmap * nchannel
  CUDA_KERNEL_LOOP(index, N) {
    int const nrow = index / nchannel;
    int const ch = index % nchannel;

    MapItype const out_map_row = d_out_map[nrow];
    MapItype const in_index = d_in_index_min[nrow];
    MapItype num_in_feat;

    if (nrow == out_nrows - 1)
      num_in_feat = nmap - in_index;
    else
      num_in_feat = d_in_index_min[nrow + 1] - in_index;
    // It is guaranteed to have at least one input per output
    MapItype curr_index, max_index = d_in_map[in_index] * nchannel + ch;
    Dtype curr_val, max_val = d_in_feat[max_index];
    for (int curr_iter = 0; curr_iter < num_in_feat; ++curr_iter) {
      curr_index = d_in_map[in_index + curr_iter] * nchannel + ch;
      curr_val = d_in_feat[curr_index];
      if (max_val < curr_val) {
        max_val = curr_val;
        max_index = curr_index;
      }
    }
    MapItype const out_ind = out_map_row * nchannel + ch;
    // TODO thrust::reduce_by_key results in erroneous results at the end for
    // very large array
    if (out_map_row < out_nrows) {
      d_out_feat[out_ind] = max_val;
      d_max_index[out_ind] = max_index;
    }
  }
}

// Put features in to the out features according to the input index.
// The input index is sorted according to the out index so no need to take out
// index
template <typename Dtype, typename Itype>
__global__ void copy_sorted(const int n, const int nrows, const int nchannel,
                            const Dtype *__restrict__ in_feat,
                            const Itype *__restrict__ in_index,
                            Dtype *__restrict__ out_feat) {
  int nrow, ch;
  CUDA_KERNEL_LOOP(index, n) {
    nrow = index / nchannel;
    ch = index % nchannel;
    out_feat[index] = in_feat[in_index[nrow] * nchannel + ch];
  }
}

namespace minkowski {

namespace detail {

template <typename Dtype>
__global__ void fill(const size_t n, Dtype *dst, Dtype const val) {
  auto const tx = threadIdx.x;
  auto const bx = blockIdx.x;
  auto const x = blockDim.x * bx + tx;
  if (x < n)
    dst[x] = val;
}

} // namespace detail

template <typename Dtype, typename MaskItype, typename MapItype>
void max_pool_forward_pointer_kernel_gpu(
    MapItype *d_in_map,     // this will be sorted
    MapItype *d_out_map,    // this will be sorted
    size_t const nmap,      // map size
    Dtype const *d_in_feat, //
    Dtype *d_out_feat,      //
    size_t const out_nrows, //
    size_t const nchannel,  //
    MaskItype *d_max_index, //
    bool const is_sorted    //
) {

  MapItype *d_scr = (MapItype *)c10::cuda::CUDACachingAllocator::raw_alloc(
      3 * (nmap + 1) * sizeof(MapItype));

  MapItype *d_index = d_scr;                          // sequence
  MapItype *d_in_map_min = d_scr + 1 * nmap + 1;      // reduced min output maps
  MapItype *d_reduced_out_map = d_scr + 2 * nmap + 2; // reduced output maps

  // create number of in_feat per out, and starting index
  thrust::sequence(thrust::device, d_index, d_index + nmap);

  ////////////////////////////////
  // Reduction
  ////////////////////////////////
  // sort d_out_map and d_in_map with the d_out_map so that in_feat are
  // placed adjacent according to out_map
  if (!is_sorted)
    thrust::sort_by_key(thrust::device, d_out_map, d_out_map + nmap, d_in_map);

  thrust::equal_to<MapItype> equal_pred;
  thrust::minimum<MapItype> min_op;

  auto reduction_pair =
      thrust::reduce_by_key(thrust::device,    // execution policy
                            d_out_map,         // key begin
                            d_out_map + nmap,  // key end
                            d_index,           // val begin
                            d_reduced_out_map, // key out begin
                            d_in_map_min,      // val out begin
                            equal_pred,        // binary pred
                            min_op);           // binary op
  CUDA_CHECK(hipStreamSynchronize(0));

  size_t num_unique_out_map = reduction_pair.first - d_reduced_out_map;

#ifdef DEBUG
  std::cout << "num_unique_out_map: " << num_unique_out_map << "\n";
  MapItype *p_scr = (MapItype *)std::malloc((nmap + 1) * 2 * sizeof(MapItype));
  CUDA_CHECK(hipMemcpy(p_scr, d_in_map_min, (nmap + 1) * 2 * sizeof(MapItype),
                        hipMemcpyDeviceToHost));
  MapItype step = std::max<MapItype>(num_unique_out_map / 100, 1);
  MapItype i = 0;
  for (; i < num_unique_out_map;) {
    std::cout << i;
    std::cout << " in_map_min: " << p_scr[i]
              << ", reduced_out_map: " << p_scr[i + 1 + nmap] << "\n";
    i += step;
  }
  i -= step;
  for (; i < num_unique_out_map; ++i) {
    std::cout << i;
    std::cout << " in_map_min: " << p_scr[i]
              << ", reduced_out_map: " << p_scr[i + 1 + nmap] << "\n";
  }
  std::free(p_scr);
  std::cout << "done printing\n";
#endif

  if (num_unique_out_map > out_nrows)
    throw std::invalid_argument(
        Formatter() << "Invalid number of out nrows: " << out_nrows
                    << ", num_unique_out_map: " << num_unique_out_map);

  // fill it with unused key
  detail::fill<<<GET_BLOCKS(out_nrows * nchannel, CUDA_NUM_THREADS),
                 CUDA_NUM_THREADS>>>(out_nrows * nchannel, d_max_index,
                                     std::numeric_limits<MaskItype>::max());

#ifdef DEBUG
  // CUDA_CHECK(hipStreamSynchronize(stream));
  std::cout << "filled\n";
#endif

  // Finally, use the max kernel to map all in_feats with the same out key to
  // out_feats Also, create out max_index for gradient
  max_pool_reduced_pointer_kernel_gpu<Dtype, MaskItype, MapItype>
      <<<GET_BLOCKS(num_unique_out_map * nchannel, CUDA_NUM_THREADS),
         CUDA_NUM_THREADS>>>(nchannel * num_unique_out_map, // N
                             num_unique_out_map, nchannel, nmap, d_in_feat,
                             d_out_feat,
                             d_max_index, // Out indices for backward
                             d_in_map,    // in index
                             d_reduced_out_map, d_in_map_min);

  // hipFree(d_in_map);
  // hipFree(d_index);
  CUDA_CHECK(hipGetLastError());
  // CUDA_CHECK(hipStreamSynchronize(stream));

  c10::cuda::CUDACachingAllocator::raw_delete((void *)d_scr);
}

template <typename Dtype, typename Itype, typename ByteAllocator>
void MaxPoolingForwardKernelGPU(
    const Dtype *d_in_feat, Dtype *d_out_feat, size_t const out_nrows,
    int *d_max_index, size_t const nchannel,
    gpu_kernel_map<Itype, ByteAllocator> const &kernel_map,
    ByteAllocator &allocator, hipStream_t stream) {

  size_t nmap = kernel_map.size();
  size_t scratch_bytes = 2 * (nmap + 1) * sizeof(Itype);
  Itype *d_scr = (Itype *)allocator.allocate(scratch_bytes);

  Itype *d_in_map = d_scr;             // all input kernel maps
  Itype *d_out_map = d_scr + nmap + 1; // all output kernel maps

  ////////////////////////////////
  // Initialize data
  ////////////////////////////////
#ifdef DEBUG
  hipMemset(d_scr, 0, scratch_bytes);
  std::cout << "out_nrows: " << out_nrows << ", nmap: " << nmap << "\n";
#endif
  Itype *d_curr_in_map = d_in_map;
  Itype *d_curr_out_map = d_out_map;

  for (auto k = kernel_map.key_cbegin(); k != kernel_map.key_cend(); ++k) {
    auto kernel_index = k->first;
    size_t curr_size = kernel_map.in_maps.size(kernel_index);
    CUDA_CHECK(hipMemcpyAsync(
        d_curr_in_map, kernel_map.in_maps.begin(kernel_index),
        curr_size * sizeof(int), hipMemcpyDeviceToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(
        d_curr_out_map, kernel_map.out_maps.begin(kernel_index),
        curr_size * sizeof(int), hipMemcpyDeviceToDevice, stream));
    d_curr_in_map += curr_size;
    d_curr_out_map += curr_size;
  }
  CUDA_CHECK(hipStreamSynchronize(stream));

  //
  max_pool_forward_pointer_kernel_gpu<Dtype, int32_t, Itype>(
      d_in_map, d_out_map, nmap, d_in_feat, d_out_feat, out_nrows, nchannel,
      d_max_index, false);

  allocator.deallocate((char *)d_scr, scratch_bytes);
}

// default_allocator
template void
MaxPoolingForwardKernelGPU<float, uint32_t, detail::default_allocator<char>>(
    const float *d_in_feat, float *d_out_feat, size_t const out_nrows,
    int32_t *d_max_index, size_t const nchannel,
    gpu_kernel_map<uint32_t, detail::default_allocator<char>> const &kernel_map,
    detail::default_allocator<char> &allocator, hipStream_t stream);

template void
MaxPoolingForwardKernelGPU<double, uint32_t, detail::default_allocator<char>>(
    const double *d_in_feat, double *d_out_feat, size_t const out_nrows,
    int32_t *d_max_index, size_t const nchannel,
    gpu_kernel_map<uint32_t, detail::default_allocator<char>> const &kernel_map,
    detail::default_allocator<char> &allocator, hipStream_t stream);

// c10_allocator
template void
MaxPoolingForwardKernelGPU<float, uint32_t, detail::c10_allocator<char>>(
    const float *d_in_feat, float *d_out_feat, size_t const out_nrows,
    int32_t *d_max_index, size_t const nchannel,
    gpu_kernel_map<uint32_t, detail::c10_allocator<char>> const &kernel_map,
    detail::c10_allocator<char> &allocator, hipStream_t stream);

template void
MaxPoolingForwardKernelGPU<double, uint32_t, detail::c10_allocator<char>>(
    const double *d_in_feat, double *d_out_feat, size_t const out_nrows,
    int32_t *d_max_index, size_t const nchannel,
    gpu_kernel_map<uint32_t, detail::c10_allocator<char>> const &kernel_map,
    detail::c10_allocator<char> &allocator, hipStream_t stream);

template <typename Dtype, typename MaskItype>
void MaxPoolingBackwardKernelGPU(Dtype *d_grad_in_feat, size_t const in_nrows,
                                 const Dtype *d_grad_out_feat,
                                 size_t const out_nrows,
                                 const MaskItype *d_max_index,
                                 size_t const nchannel) {
  size_t const num_kernels = out_nrows * nchannel;
  // Assume that gradients for input feature are all set to zero
  LOG_DEBUG("MaxPool Backward GPU with #", num_kernels, out_nrows, nchannel);
  set_gradient<Dtype, MaskItype>
      <<<GET_BLOCKS(num_kernels, CUDA_NUM_THREADS), CUDA_NUM_THREADS>>>(
          num_kernels, d_grad_out_feat, d_grad_in_feat, d_max_index,
          std::numeric_limits<MaskItype>::max());

  // CUDA_CHECK(hipStreamSynchronize(stream));
}

template void max_pool_forward_pointer_kernel_gpu<float, int32_t, uint32_t>(
    uint32_t *d_in_map,     //
    uint32_t *d_out_map,    //
    size_t const nmap,      //
    float const *d_in_feat, //
    float *d_out_feat,      //
    size_t const out_nrows, //
    size_t const nchannel,  //
    int32_t *d_max_index,   //
    bool const is_sorted    //
);

template void max_pool_forward_pointer_kernel_gpu<double, int32_t, uint32_t>(
    uint32_t *d_in_map,      //
    uint32_t *d_out_map,     //
    size_t const nmap,       //
    double const *d_in_feat, //
    double *d_out_feat,      //
    size_t const out_nrows,  //
    size_t const nchannel,   //
    int32_t *d_max_index,    //
    bool const is_sorted     //
);

template void max_pool_forward_pointer_kernel_gpu<float, int32_t, int32_t>(
    int32_t *d_in_map,      //
    int32_t *d_out_map,     //
    size_t const nmap,      //
    float const *d_in_feat, //
    float *d_out_feat,      //
    size_t const out_nrows, //
    size_t const nchannel,  //
    int32_t *d_max_index,   //
    bool const is_sorted    //
);

template void max_pool_forward_pointer_kernel_gpu<double, int32_t, int32_t>(
    int32_t *d_in_map,       //
    int32_t *d_out_map,      //
    size_t const nmap,       //
    double const *d_in_feat, //
    double *d_out_feat,      //
    size_t const out_nrows,  //
    size_t const nchannel,   //
    int32_t *d_max_index,    //
    bool const is_sorted     //
);

template void MaxPoolingBackwardKernelGPU<float, int32_t>(
    float *d_grad_in_feat, size_t const in_nrows, const float *d_grad_out_feat,
    size_t const out_nrows, const int32_t *d_max_index, size_t const nchannel);

template void MaxPoolingBackwardKernelGPU<double, int32_t>(
    double *d_grad_in_feat, size_t const in_nrows,
    const double *d_grad_out_feat, size_t const out_nrows,
    const int32_t *d_max_index, size_t const nchannel);

// int64
template void max_pool_forward_pointer_kernel_gpu<float, int64_t, int64_t>(
    int64_t *d_in_map,      //
    int64_t *d_out_map,     //
    size_t const nmap,      //
    float const *d_in_feat, //
    float *d_out_feat,      //
    size_t const out_nrows, //
    size_t const nchannel,  //
    int64_t *d_max_index,   //
    bool const is_sorted    //
);

template void max_pool_forward_pointer_kernel_gpu<double, int64_t, int64_t>(
    int64_t *d_in_map,       //
    int64_t *d_out_map,      //
    size_t const nmap,       //
    double const *d_in_feat, //
    double *d_out_feat,      //
    size_t const out_nrows,  //
    size_t const nchannel,   //
    int64_t *d_max_index,    //
    bool const is_sorted     //
);

template void MaxPoolingBackwardKernelGPU<float, int64_t>(
    float *d_grad_in_feat, size_t const in_nrows, const float *d_grad_out_feat,
    size_t const out_nrows, const int64_t *d_max_index, size_t const nchannel);

template void MaxPoolingBackwardKernelGPU<double, int64_t>(
    double *d_grad_in_feat, size_t const in_nrows,
    const double *d_grad_out_feat, size_t const out_nrows,
    const int64_t *d_max_index, size_t const nchannel);

} // end namespace minkowski

#endif // GPU_POOLING_MAX_KERNEL
