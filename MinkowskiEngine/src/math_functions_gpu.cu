#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA Corporation.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "allocators.cuh"
#include "math_functions.cuh"

namespace minkowski {

// CUBLAS, CUSPARSE assume all dense matrices to be col major
template <>
void gpu_gemm<float>(hipblasHandle_t handle, const CBLAS_TRANSPOSE TransA,
                     const CBLAS_TRANSPOSE TransB, const int M, const int N,
                     const int K, const float alpha, const float *A,
                     const float *B, const float beta, float *C) {
  // Note that cublas follows (column-major) fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, N));
}

template <>
void gpu_gemm<double>(hipblasHandle_t handle, const CBLAS_TRANSPOSE TransA,
                      const CBLAS_TRANSPOSE TransB, const int M, const int N,
                      const int K, const double alpha, const double *A,
                      const double *B, const double beta, double *C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(handle, cuTransB, cuTransA, N, M, K, &alpha, B, ldb,
                           A, lda, &beta, C, N));
}

template <typename Dtype>
__global__ void addition_kernel(const int n, const Dtype *a, const Dtype *b,
                                Dtype *y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = a[index] + b[index]; }
}

template <typename Dtype>
__global__ void multiplication_kernel(const int n, const Dtype *a,
                                      const Dtype *b, Dtype *y) {
  CUDA_KERNEL_LOOP(index, n) { y[index] = a[index] * b[index]; }
}

template <typename Dtype>
void gpu_addition(const int N, const Dtype *a, const Dtype *b, Dtype *y,
                  hipStream_t stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  addition_kernel<Dtype>
      <<<GET_BLOCKS(N, CUDA_NUM_THREADS), CUDA_NUM_THREADS, 0, stream>>>(N, a,
                                                                         b, y);
}

template void gpu_addition<float>(const int N, const float *a, const float *b,
                                  float *y, hipStream_t stream);

template void gpu_addition<double>(const int N, const double *a,
                                   const double *b, double *y,
                                   hipStream_t stream);

template <typename Dtype>
void gpu_multiplication(const int N, const Dtype *a, const Dtype *b, Dtype *y,
                        hipStream_t stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  multiplication_kernel<Dtype>
      <<<GET_BLOCKS(N, CUDA_NUM_THREADS), CUDA_NUM_THREADS, 0, stream>>>(N, a,
                                                                         b, y);
}

template void gpu_multiplication<float>(const int N, const float *a,
                                        const float *b, float *y,
                                        hipStream_t stream);

template void gpu_multiplication<double>(const int N, const double *a,
                                         const double *b, double *y,
                                         hipStream_t stream);

template <typename Dtype>
__global__ void col2row_major_kernel(const int n, const int nrows,
                                     const int ncols, const Dtype *colA,
                                     Dtype *rowA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index % nrows;
    j = index / nrows;
    rowA[i * ncols + j] = colA[index];
  }
}

template <typename Dtype>
void col2row_major(const int nrows, const int ncols, const Dtype *colA,
                   Dtype *rowA, hipStream_t stream) {
  col2row_major_kernel<Dtype>
      <<<GET_BLOCKS(nrows * ncols, CUDA_NUM_THREADS), CUDA_NUM_THREADS, 0,
         stream>>>(nrows * ncols, nrows, ncols, colA, rowA);
}

template void col2row_major<float>(const int nrows, const int ncols,
                                   const float *colA, float *rowA,
                                   hipStream_t stream);

template void col2row_major<double>(const int nrows, const int ncols,
                                    const double *colA, double *rowA,
                                    hipStream_t stream);

template <typename Dtype>
__global__ void row2col_major_kernel(const int n, const int nrows,
                                     const int ncols, const Dtype *rowA,
                                     Dtype *colA) {
  int i, j;
  CUDA_KERNEL_LOOP(index, n) {
    i = index / ncols;
    j = index % ncols;
    colA[i + j * nrows] = rowA[index];
  }
}

template <typename Dtype>
void row2col_major(const int nrows, const int ncols, const Dtype *colA,
                   Dtype *rowA, hipStream_t stream) {
  row2col_major_kernel<Dtype>
      <<<GET_BLOCKS(nrows * ncols, CUDA_NUM_THREADS), CUDA_NUM_THREADS, 0,
         stream>>>(nrows * ncols, nrows, ncols, colA, rowA);
}

template void row2col_major<float>(const int nrows, const int ncols,
                                   const float *colA, float *rowA,
                                   hipStream_t stream);

template void row2col_major<double>(const int nrows, const int ncols,
                                    const double *colA, double *rowA,
                                    hipStream_t stream);

// Sort (row, col) pairs row-major order.
template <typename allocator_type>
void sort_coo_gpu(hipsparseHandle_t handle, const int m, const int n,
                  const int nnz, int *d_coo_row, int *d_coo_col,
                  allocator_type &allocator) {
  size_t pBufferSizeInBytes = 0;
  void *pBuffer = NULL;
  int *P = NULL;

  // step 1: allocate buffer
  CUSPARSE_CHECK(hipsparseXcoosort_bufferSizeExt(
      handle, m, n, nnz, d_coo_row, d_coo_col, &pBufferSizeInBytes));
  pBuffer = (void *)allocator.allocate(sizeof(char) * pBufferSizeInBytes);
  // step 2: setup permutation vector P to identity
  P = (int *)allocator.allocate(sizeof(int) * nnz);
  CUSPARSE_CHECK(hipsparseCreateIdentityPermutation(handle, nnz, P));
  // step 3: sort COO
  CUSPARSE_CHECK(hipsparseXcoosortByRow(handle, m, n, nnz, d_coo_row, d_coo_col,
                                       P, pBuffer));
  allocator.deallocate((char *)pBuffer, sizeof(char) * pBufferSizeInBytes);
  allocator.deallocate((char *)P, sizeof(int) * nnz);
}

template void sort_coo_gpu<detail::default_allocator<char>>(
    hipsparseHandle_t handle, const int m, const int n, const int nnz,
    int *d_coo_row, int *d_coo_col, detail::default_allocator<char> &allocator);

template void sort_coo_gpu<detail::c10_allocator<char>>(
    hipsparseHandle_t handle, const int m, const int n, const int nnz,
    int *d_coo_row, int *d_coo_col, detail::c10_allocator<char> &allocator);

} // end namespace minkowski
