#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020 NVIDIA Corporation.
 * Copyright (c) 2018-2020 Chris Choy (chrischoy@ai.stanford.edu).
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 *
 * Please cite "4D Spatio-Temporal ConvNets: Minkowski Convolutional Neural
 * Networks", CVPR'19 (https://arxiv.org/abs/1904.08755) if you use any part
 * of the code.
 */
#include "coordinate_map.hpp"
#include "coordinate_map_key.hpp"
#include "coordinate_map_manager.hpp"
#include "errors.hpp"
#include "types.hpp"
#include "utils.hpp"

#include "convolution_kernel.cuh"
#include "kernel_map.cuh"

// Ninja
#include "convolution_transpose_cpu.cpp"

#include <ATen/cuda/CUDAUtils.h>
#include <pybind11/pybind11.h>
#include <torch/extension.h>

namespace minkowski {

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
at::Tensor ConvolutionTransposeForwardGPU(
    at::Tensor const &in_feat,                         //
    at::Tensor const &kernel,                          //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    bool generate_new_coordinates,                     //
    ConvolutionMode::Type const convolution_mode,      //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<coordinate_type, TemplatedAllocator> *p_map_manager) {

  ASSERT(in_feat.is_contiguous(), "in_feat must be contiguous");
  ASSERT(kernel.is_contiguous(), "kernel must be contiguous");

  ASSERT(in_feat.is_cuda(), "in_feat must be CUDA");
  ASSERT(kernel.is_cuda(), "kernel must be CUDA");
  ASSERT(at::cuda::check_device({in_feat, kernel}),
         "in_feat and kernel must be on the same device");

  ASSERT(in_feat.scalar_type() == kernel.scalar_type(), "type mismatch");

  ASSERT(in_feat.dim() == 2, "in_feat.dim():", in_feat.dim());
  ASSERT(kernel.dim() == 3, "kernel.dim():", kernel.dim());

  ASSERT(in_feat.size(1) == kernel.size(1),
         "Input feature size and kernel size mismatch");

  // TODO kernel volume assertion.

  // create out coordinate map
  // TODO: custom upsampling
  coordinate_map_key_type in_key = p_in_map_key->get_key();
  ASSERT(p_map_manager->exists(in_key), ERROR_MAP_NOT_FOUND);

  ASSERT(in_feat.size(0) == p_map_manager->size(in_key), "Invalid in_feat size",
         in_feat.size(0), "!=", p_map_manager->size(in_key));

  if (!p_out_map_key->is_key_set()) {
    auto map_it = p_map_manager->find(p_in_map_key->get_key());
    ASSERT(map_it != p_map_manager->map_end(), ERROR_MAP_NOT_FOUND);
    auto const &in_map = (*map_it).second;

    auto out_tensor_stride = detail::stride_tensor_stride(
        in_map.get_tensor_stride(), kernel_stride, true /* is_transpose */);
    auto kernel_region = cpu_kernel_region<coordinate_type>(
        region_type,              //
        in_map.coordinate_size(), //
        out_tensor_stride.data(), //
        kernel_size.data(),       //
        kernel_dilation.data(),   //
        0,                        // volume
        offset.data_ptr<coordinate_type>(), offset.size(0),
        true // is_transpose
    );

    coordinate_map_key_type out_key = std::get<0>(p_map_manager->stride_region(
        in_key, kernel_region, out_tensor_stride, generate_new_coordinates));
    LOG_DEBUG("ConvolutionTranspose out key:", out_key);
    p_out_map_key->set_key(out_key);
  }

  auto const &in_out = p_map_manager->kernel_map(p_in_map_key,            //
                                                 p_out_map_key,           //
                                                 kernel_size,             //
                                                 kernel_stride,           //
                                                 kernel_dilation,         //
                                                 region_type,             //
                                                 offset,                  //
                                                 true /* is_transpose */, //
                                                 false /* is_pool */);

#ifdef DEBUG
  LOG_DEBUG("Transposed kernel map in_maps:",
            in_out.out_maps.begin() - in_out.in_maps.begin());
#endif

  auto const out_nrows = p_map_manager->size(p_out_map_key->get_key());
  at::Tensor out_feat =
      torch::zeros({out_nrows, kernel.size(2)}, in_feat.options());
  LOG_DEBUG("In feat:", in_feat.size(0), "x", in_feat.size(1), "-> out feat",
            out_feat.size(0), "x", out_feat.size(1));

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  AT_DISPATCH_FLOATING_TYPES(
      in_feat.scalar_type(), "convolution_transpose_forward_gpu", [&] {
        TemplatedAllocator<char> byte_allocator;
        ConvolutionForwardKernelGPU<scalar_t, default_types::index_type,
                                    TemplatedAllocator<char>>(
            in_feat.template data_ptr<scalar_t>(),  //
            in_feat.size(1),                        //
            out_feat.template data_ptr<scalar_t>(), //
            out_feat.size(1),                       //
            kernel.template data_ptr<scalar_t>(),   //
            in_out,                                 //
            in_feat.size(0),                        //
            out_nrows,                              //
            byte_allocator,                         //
            p_map_manager->algorithm(),             //
            convolution_mode, handle, stream);
      });

  return out_feat;
}

template <typename coordinate_type,
          template <typename C> class TemplatedAllocator>
std::pair<at::Tensor, at::Tensor> ConvolutionTransposeBackwardGPU(
    at::Tensor const &in_feat,                         //
    at::Tensor const &grad_out_feat,                   //
    at::Tensor const &kernel,                          //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    ConvolutionMode::Type const convolution_mode,      //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<coordinate_type, TemplatedAllocator> *p_map_manager) {

  ASSERT(in_feat.is_contiguous(), "in_feat must be contiguous");
  ASSERT(grad_out_feat.is_contiguous(), "grad_out_feat must be contiguous");
  ASSERT(kernel.is_contiguous(), "kernel must be contiguous");

  ASSERT(in_feat.is_cuda(), "in_feat must be CUDA");
  ASSERT(grad_out_feat.is_cuda(), "in_feat must be CUDA");
  ASSERT(kernel.is_cuda(), "kernel must be CUDA");
  ASSERT(at::cuda::check_device({in_feat, grad_out_feat, kernel}),
         "in_feat, grad_out_feat, kernel must be on the same device");

  ASSERT(in_feat.scalar_type() == kernel.scalar_type(), "type mismatch");
  ASSERT(in_feat.scalar_type() == grad_out_feat.scalar_type(), "type mismatch");

  ASSERT(in_feat.dim() == 2, "in_feat.dim():", in_feat.dim());
  ASSERT(grad_out_feat.dim() == 2, "grad_out_feat.dim():", grad_out_feat.dim());
  ASSERT(kernel.dim() == 3, "kernel.dim():", kernel.dim());

  ASSERT(in_feat.size(1) == kernel.size(1),
         "Input feature size and kernel size mismatch");

  coordinate_map_key_type in_key = p_in_map_key->get_key();
  ASSERT(p_map_manager->exists(in_key), ERROR_MAP_NOT_FOUND);
  coordinate_map_key_type out_key = p_out_map_key->get_key();
  ASSERT(p_map_manager->exists(out_key), ERROR_MAP_NOT_FOUND);

  auto const &in_out = p_map_manager->kernel_map(
      p_in_map_key,    //
      p_out_map_key,   //
      kernel_size,     //
      kernel_stride,   //
      kernel_dilation, //
      region_type,     //
      offset, true /* is_transpose */, false /* is_pool */);

  at::Tensor grad_in_feat =
      torch::zeros({in_feat.size(0), in_feat.size(1)}, in_feat.options());
  at::Tensor grad_kernel = torch::zeros(
      {kernel.size(0), kernel.size(1), kernel.size(2)}, kernel.options());

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  AT_DISPATCH_FLOATING_TYPES(
      in_feat.scalar_type(), "convolution_transpose_backward_gpu", [&] {
        TemplatedAllocator<char> byte_allocator;
        ConvolutionBackwardKernelGPU<scalar_t, default_types::index_type,
                                     TemplatedAllocator<char>>(
            in_feat.template data_ptr<scalar_t>(),       //
            grad_in_feat.template data_ptr<scalar_t>(),  //
            in_feat.size(1),                             //
            grad_out_feat.template data_ptr<scalar_t>(), //
            grad_out_feat.size(1),                       //
            kernel.template data_ptr<scalar_t>(),        //
            grad_kernel.template data_ptr<scalar_t>(),   //
            in_out,                                      //
            in_feat.size(0),                             //
            grad_out_feat.size(0),                       //
            byte_allocator,                              //
            p_map_manager->algorithm(),                  //
            convolution_mode,                            //
            handle, stream);
      });

  return std::make_pair(grad_in_feat, grad_kernel);
}

// Forward
// default_allocator
template at::Tensor
ConvolutionTransposeForwardGPU<default_types::dcoordinate_type,
                               detail::default_allocator>(
    at::Tensor const &in_feat,                         //
    at::Tensor const &kernel,                          //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    bool generate_new_coordinates,                     //
    ConvolutionMode::Type const convolution_mode,      //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::default_allocator>
        *p_map_manager);

// c10_allocator
template at::Tensor
ConvolutionTransposeForwardGPU<default_types::dcoordinate_type,
                               detail::c10_allocator>(
    at::Tensor const &in_feat,                         //
    at::Tensor const &kernel,                          //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    bool generate_new_coordinates,                     //
    ConvolutionMode::Type const convolution_mode,      //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::c10_allocator>
        *p_map_manager);

// Backward
// default_allocator
template std::pair<at::Tensor, at::Tensor>
ConvolutionTransposeBackwardGPU<default_types::dcoordinate_type,
                                detail::default_allocator>(
    at::Tensor const &in_feat,                         //
    at::Tensor const &grad_out_feat,                   //
    at::Tensor const &kernel,                          //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    ConvolutionMode::Type const convolution_mode,      //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::default_allocator>
        *p_map_manager);

// c10_allocator
template std::pair<at::Tensor, at::Tensor>
ConvolutionTransposeBackwardGPU<default_types::dcoordinate_type,
                                detail::c10_allocator>(
    at::Tensor const &in_feat,                         //
    at::Tensor const &grad_out_feat,                   //
    at::Tensor const &kernel,                          //
    default_types::stride_type const &kernel_size,     //
    default_types::stride_type const &kernel_stride,   //
    default_types::stride_type const &kernel_dilation, //
    RegionType::Type const region_type,                //
    at::Tensor const &offset,                          //
    ConvolutionMode::Type const convolution_mode,      //
    CoordinateMapKey *p_in_map_key,                    //
    CoordinateMapKey *p_out_map_key,                   //
    gpu_manager_type<default_types::dcoordinate_type, detail::c10_allocator>
        *p_map_manager);

} // end namespace minkowski
